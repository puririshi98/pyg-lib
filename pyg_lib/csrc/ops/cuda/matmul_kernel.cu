#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#include <cutlass/util/host_tensor.h>
#include <torch/library.h>
#include <torch/nn/functional/padding.h>

#include "pyg_lib/csrc/utils/convert.h"

namespace pyg {
namespace ops {

namespace {
namespace F = torch::nn::functional;
using namespace torch::indexing;

at::Tensor pad_to_align(const at::Tensor& input) {
  std::cout << "================= input.shape =================" << std::endl;
  std::cout << input.size(-2);
  std::cout << ",";
  std::cout << input.size(-1) << std::endl;
  int dim_0_pad = (ceil(input.size(-2) / 4) * 4) - input.size(-2);
  int dim_1_pad = (ceil(input.size(-1) / 4) * 4) - input.size(-1);
  std::cout << "================= pads =================" << std::endl;
  std::cout << dim_0_pad;
  std::cout << ",";
  std::cout << dim_1_pad << std::endl;

  return F::pad(
      input,
      F::PadFuncOptions({0, dim_1_pad, 0, dim_0_pad}).mode(torch::kConstant));
}

void grouped_matmul_out_kernel(const std::vector<at::Tensor>& input,
                               const std::vector<at::Tensor>& other,
                               const std::vector<at::Tensor>& out) {
  // TODO (matthias) Check tensor devices.

  const auto num_matrices = input.size();
  std::vector<at::Tensor> new_input, new_other, new_out;

  // TODO (matthias) Allow for other types than `float`.
  // TODO (matthias) Are these attributes correctly set?
  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
      float,                             // Element A
      cutlass::layout::RowMajor,         // Layout A
      cutlass::ComplexTransform::kNone,  //
      4,                          // Granularity A (4 is the max for 32 bit)
      float,                      // Element B
      cutlass::layout::RowMajor,  // Layout B
      cutlass::ComplexTransform::kNone,              //
      4,                                             // Granularity B
      float,                                         // Element C&D
      cutlass::layout::RowMajor,                     // Layout C&D
      float,                                         // Element Accumulator
      cutlass::arch::OpClassTensorOp,                // Operator Class Tag
      cutlass::arch::Sm80,                           // Architecture
      cutlass::gemm::GemmShape<256, 128, 32>,        // Threadblock-level Tile
      cutlass::gemm::GemmShape<64, 64, 32>,          // Warp-level Tile
      cutlass::gemm::GemmShape<16, 8, 8>,            // Warp-level Tile
      cutlass::epilogue::thread::LinearCombination<  // Epilogue
          float, 4, float, float>,                   //
      cutlass::gemm::threadblock::                   // Swizzling Operator
      GemmIdentityThreadblockSwizzle<8>,             //
      3,                                             // Stages
      cutlass::arch::OpMultiplyAdd                   // Operation
      >::GemmKernel;

  std::vector<float*> ptr_A_host(num_matrices);
  std::vector<float*> ptr_B_host(num_matrices);
  std::vector<float*> ptr_C_host(num_matrices);

  for (size_t i = 0; i < num_matrices; ++i) {
    if (input[i].size(-1) % 4 != 0 || input[i].size(-2) % 4 != 0) {
      new_input.push_back(pad_to_align(input[i]).contiguous());
    } else {
      new_input.push_back(input[i].contiguous());
    }
    ptr_A_host[i] = new_input[i].data_ptr<float>();
    if (other[i].size(-1) % 4 != 0 || other[i].size(-2) % 4 != 0) {
      new_other.push_back(pad_to_align(other[i]).contiguous());
    } else {
      new_other.push_back(other[i].contiguous());
    }
    ptr_B_host[i] = new_other[i].data_ptr<float>();
    if (out[i].size(-1) % 4 != 0 || out[i].size(-2) % 4 != 0) {
      new_out.push_back(pad_to_align(out[i]).contiguous());
    } else {
      new_out.push_back(out[i].contiguous());
    }
    ptr_C_host[i] = new_out[i].data_ptr<float>();
  }

  cutlass::DeviceAllocation<float*> ptr_A;
  ptr_A.reset(num_matrices);
  ptr_A.copy_from_host(ptr_A_host.data());

  cutlass::DeviceAllocation<float*> ptr_B;
  ptr_B.reset(num_matrices);
  ptr_B.copy_from_host(ptr_B_host.data());

  cutlass::DeviceAllocation<float*> ptr_C;
  ptr_C.reset(num_matrices);
  ptr_C.copy_from_host(ptr_C_host.data());

  std::vector<cutlass::gemm::GemmCoord> all_problems(num_matrices);
  std::vector<int64_t> ld_A_host(num_matrices);
  std::vector<int64_t> ld_B_host(num_matrices);
  std::vector<int64_t> ld_C_host(num_matrices);
  for (size_t i = 0; i < num_matrices; ++i) {
    auto m = new_input[i].size(0), k = new_input[i].size(1),
         n = new_out[i].size(1);

    TORCH_CHECK(new_input[i].size(-1) == new_other[i].size(-2),
                "Shape mismatch");
    all_problems[i] = cutlass::gemm::GemmCoord(m, n, k);
    ld_A_host[i] = GemmKernel::LayoutA::packed({m, k}).stride(0);
    ld_B_host[i] = GemmKernel::LayoutB::packed({k, n}).stride(0);
    ld_C_host[i] = GemmKernel::LayoutC::packed({m, n}).stride(0);
  }

  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> all_problems_device;
  all_problems_device.reset(num_matrices);
  all_problems_device.copy_from_host(all_problems.data());

  cutlass::DeviceAllocation<int64_t> ld_A;
  ld_A.reset(num_matrices);
  ld_A.copy_from_host(ld_A_host.data());

  cutlass::DeviceAllocation<int64_t> ld_B;
  ld_B.reset(num_matrices);
  ld_B.copy_from_host(ld_B_host.data());

  cutlass::DeviceAllocation<int64_t> ld_C;
  ld_C.reset(num_matrices);
  ld_C.copy_from_host(ld_C_host.data());

  using EpilogueOutputOp = typename GemmKernel::Epilogue::OutputOp;
  typename EpilogueOutputOp::Params epilogue_op(1.0, 0.0);

  using GemmGrouped = cutlass::gemm::device::GemmGrouped<GemmKernel>;
  typename GemmGrouped::Arguments args(
      all_problems_device.get(), num_matrices, /*threadblock_count=*/1024,
      epilogue_op, ptr_A.get(), ptr_B.get(), ptr_C.get(), ptr_C.get(),
      ld_A.get(), ld_B.get(), ld_C.get(), ld_C.get());

  GemmGrouped gemm;
  auto status = gemm.initialize(args);
  TORCH_CHECK(status == cutlass::Status::kSuccess, "GroupedGEMM init failed");
  status = gemm.run();
  TORCH_CHECK(status == cutlass::Status::kSuccess, "GroupedGEMM run failed");
  for (size_t i = 0; i < num_matrices; ++i) {
    std::cout << "================= out.shape =================" << std::endl;
    std::cout << out[i].size(0);
    std::cout << ",";
    std::cout << out[i].size(1) << std::endl;
    std::cout << "================= new_out.shape ================="
              << std::endl;
    std::cout << new_out[i].size(0);
    std::cout << ",";
    std::cout << new_out[i].size(1) << std::endl;
    out[i].index_put_({None}, new_out[i].index({Slice(None, out[i].size(0)),
                                                Slice(None, out[i].size(1))}));
  }
}

std::vector<at::Tensor> grouped_matmul_kernel(
    const std::vector<at::Tensor>& input,
    const std::vector<at::Tensor>& other) {
  std::vector<at::Tensor> out(input.size());
  for (size_t i = 0; i < input.size(); ++i)
    out[i] = input[i].new_empty({input[i].size(0), other[i].size(-1)});
  grouped_matmul_out_kernel(input, other, out);

  return out;
}

at::Tensor segment_matmul_kernel(const at::Tensor& input,
                                 const at::Tensor& ptr,
                                 const at::Tensor& other) {
  const auto size = pyg::utils::size_from_ptr(ptr).cpu();
  // TODO (matthias) Allow for other types than `int64_t`.
  const auto sizes = at::IntArrayRef(size.data_ptr<int64_t>(), size.numel());
  const auto out = input.new_empty({input.size(0), other.size(-1)});

  // TODO (matthias) Better handle non-contiguous memory layouts.
  grouped_matmul_out_kernel(
      input.contiguous().split_with_sizes(/*split_size=*/sizes, /*dim=*/0),
      other.contiguous().split(/*split_size=*/1, /*dim=*/0),
      out.split_with_sizes(/*split_size=*/sizes, /*dim=*/0));

  return out;
}

}  // namespace

TORCH_LIBRARY(pyg, m) {
  m.def("pyg::cuda_grouped_matmul(Tensor[] input, Tensor[] other) -> Tensor[]");
  m.def(
      "pyg::cuda_segment_matmul(Tensor input, Tensor ptr, Tensor other) -> "
      "Tensor");
}

TORCH_LIBRARY_IMPL(pyg, CUDA, m) {
  m.impl(TORCH_SELECTIVE_NAME("pyg::cuda_grouped_matmul"),
         TORCH_FN(grouped_matmul_kernel));
  m.impl(TORCH_SELECTIVE_NAME("pyg::cuda_segment_matmul"),
         TORCH_FN(segment_matmul_kernel));
}

}  // namespace ops
}  // namespace pyg
